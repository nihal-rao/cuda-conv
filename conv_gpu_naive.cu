#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "conv_gpu.cuh"

__global__ void convGpuNaive(const float *in, float *out, int H_in, int W_in, int K)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int H_out = H_in - K + 1;
    int W_out = W_in - K + 1;
    float sum = 0.0;
    int row_lim = min(H_out, blockDim.y*blockIdx.y);
    int col_lim = min(W_out, blockDim.x*blockIdx.x);

    if (row < H_out && col < W_out)
    {
        for (int i = 0; i < K; i++)
        {
            for (int j = 0; j < K; j++)
            {
                sum += d_mask[i * K + j] *in[(i+row) * W_in + j+col];
            }
        }
        out[row*W_out+col]=sum;
    }
}
