#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "conv_gpu.cuh"

__global__ void convGpuTiled(const float *in, float *out, int H_in, int W_in, int K, int TILE_SIZE)
{
    __shared__ float tile[32 * 32];
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    int H_out = H_in - K + 1;
    int W_out = W_in - K + 1;
    float sum = 0.0;

    if(row<H_in && col<W_in)
    {
        tile[threadIdx.y*32+threadIdx.x] = in[row*W_in+col];
    }
    __syncthreads();

    if(threadIdx.y<TILE_SIZE && threadIdx.x<TILE_SIZE && row < H_out && col < W_out)
    {   
        for (int i = 0; i < K; i++)
        {
            for (int j = 0; j < K; j++)
            {
                sum += d_mask[i * K + j] *tile[(i + threadIdx.y) * 32 + j + threadIdx.x];
            }
        }
        out[row*W_out+col]=sum;
    }
}
