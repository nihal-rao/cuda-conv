#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "conv_gpu.cuh"

__global__ void convGpuTiled(const float *in, float *out, int H_in, int W_in, int K)
{
    __shared__ float tile[32 * 32];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int H_out = H_in - K + 1;
    int W_out = W_in - K + 1;
    float sum = 0.0;
    int row_lim = min(H_out, blockDim.y*(blockIdx.y+1));
    int col_lim = min(W_out, blockDim.x*(blockIdx.x+1));

    if (row < H_out && col < W_out)
    {
        tile[threadIdx.y * blockDim.x + threadIdx.x] = in[row * W_in + col];
        __syncthreads();
        for (int i = 0; i < K; i++)
        {
            for (int j = 0; j < K; j++)
            {
                if (i + row >= row_lim || j + col >= col_lim)
                {
                    sum += d_mask[i * K + j] *in[(i+row) * W_in + j+col];
                }
                else
                {
                    sum += d_mask[i * K + j] *tile[(i + threadIdx.y) * blockDim.x + j + threadIdx.x]; //
                }
            }
        }
        out[row*W_out+col]=sum;
    }
}
