#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include "conv.h"
#include "conv_gpu.cuh"
#include <hip/hip_runtime.h>

#define THRESHOLD (0.001)

using std::cout;
using std::endl;
using std::chrono::steady_clock;

__constant__ float d_mask[256];
__constant__ float debug;

void check_result(const float* w_ref, const float* w_opt, int h, int w) 
{
  float maxdiff = 0.0, this_diff = 0.0;
  int numdiffs = 0;

  for (int i = 0; i < h; i++) {
    for (int j = 0; j < w; j++) {
        this_diff = w_ref[i*w+j] - w_opt[i*w+j];
        if (std::fabs(this_diff) > THRESHOLD) {
          numdiffs++;
          if (this_diff > maxdiff) {
            maxdiff = this_diff;
          }
        }
      }
    }

  if (numdiffs > 0) {
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD << "; Max Diff = " << maxdiff
         << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

void display_result(const float* w_opt, int h, int w) 
{

  for (int i = 0; i < h; i++) {
    for (int j = 0; j < w; j++) {
        cout<<w_opt[i*w+j]<<"  ";
      }
    cout<<endl;
    }
}

int main(int argc, char* argv[])
{
    int H_in = std::strtol(argv[1], nullptr, 0);
    int W_in = std::strtol(argv[2], nullptr, 0);
    int K = std::strtol(argv[3], nullptr, 0);

    int H_out = H_in-K+1;
    int W_out = W_in-K+1;
    float * in, *out, *mask, *h_out_gpu;
    in = (float*)malloc(sizeof(float)*H_in*W_in);
    out = (float*)malloc(sizeof(float)*H_out*W_out);
    h_out_gpu = (float*)malloc(sizeof(float)*H_out*W_out);
    mask = (float*)malloc(sizeof(float)*K*K);

    for(int i=0;i<H_in;i++)
    {
        for(int j=0;j<W_in;j++)
        {
            in[i*W_in+j]=rand()%256;
            // in[i*W_in+j]=1;
        }
    }

    for(int i=0;i<H_out;i++)
    {
        for(int j=0;j<W_out;j++)
        {
            out[i*W_out+j]=0.0;
        }
    }
    
    for(int i=0;i<K;i++)
    {
        for(int j=0;j<K;j++)
        {
        float scale = rand() / (float) RAND_MAX;
        mask[i*K+j]= -1 + scale * (2);
        // mask[i*K+j]=rand()%3;
        // mask[i*K+j]=1;
        }
    }

    auto start_cpu = steady_clock::now();
    convCpu(in, out, mask, H_in, W_in, K);
    auto end_cpu = steady_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu).count();
    cout<<" CPU time is "<<duration<<" ms"<<endl;
    // cout<<" CPU result is "<<endl;
    // display_result(out, H_out, W_out);
    // display_result(in, H_in, W_in);

    float *d_in, *d_out;
    hipMalloc(&d_in, sizeof(float)*H_in*W_in);
    hipMalloc(&d_out, sizeof(double)*H_out*W_out);
    hipMemcpy(d_in, in, sizeof(float)*H_in*W_in, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask, sizeof(float)*K*K);

    int block_size_h = 32;
    int block_size_w = 32;
    
    dim3 threadsPerBlock(block_size_w, block_size_h);
    dim3 numBlocks((int)ceil(W_in/(float)block_size_w), (int)ceil(H_in/(float)block_size_h));

    hipEvent_t start, stop;
    float gpu_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // convGpuTiled<<<numBlocks, threadsPerBlock>>>(d_in, d_out, H_in, W_in, K);
    convGpuNaive<<<numBlocks, threadsPerBlock>>>(d_in, d_out, H_in, W_in, K);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&gpu_time, start, stop);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    cout << "GPU Time is "<<gpu_time << " ms" << endl;

    hipMemcpy(h_out_gpu, d_out, sizeof(float)*H_out*W_out, hipMemcpyDeviceToHost);
    cout<<" GPU result is "<<endl;
    // display_result(h_out_gpu, H_out, W_out);
    check_result(out, h_out_gpu, H_out, W_out);

    hipFree(d_in);
    hipFree(d_out);
    
    free(in);
    free(out);
    free(mask);
    free(h_out_gpu);
}